#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "png_util.h"
#include <hip/hip_runtime.h>
#define min(X,Y) ((X) < (Y) ? (X) : (Y))
#define max(X,Y) ((X) > (Y) ? (X) : (Y))

#define W 500
#define H 500


#define CUDA_CALL(x) {hipError_t cuda_error__ = (x); if (cuda_error__) printf("CUDA error: " #x " returned \"%s\"\n", hipGetErrorString(cuda_error__));}

__global__ void calcAccel(double dx2inv, double dy2inv, int nx, int ny, double *d_z, double *d_a){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    
    int P = i + j*nx;           
    int n = i + (j+1)*nx;       
    int s = i + (j-1)*nx;       
    int e = (i+1) + j*nx;      
    int w = (i-1) + j*nx; 

    if (i > 0 && i < nx-1 && j > 0 && j < ny-1){
        d_a[P] = dx2inv*(d_z[s] + d_z[n] - 2.0*d_z[P]) + dy2inv*(d_z[w] + d_z[e] - 2.0*d_z[P]);
d_a[P] = 0.5*d_a[P];
    }
}

__global__ void calcVelAndPos(double dt, int nx, int ny, double *d_z, double *d_v, double *d_a){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    int P = i + j*nx;

    d_v[P] = d_v[P] + dt*d_a[P];
    d_z[P] = d_z[P] + dt*d_v[P];
}

int main(int argc, char ** argv) {
    int nx = W;
    int ny = H;
    int nt = 10000; 
    int frame=0;
    //int nt = 1000000;
    int r,c,it;
    double *d_z, *d_a, *d_v;
    double dx,dy,dt;
    double max,min;
    double tmax;
    double dx2inv, dy2inv;
    char filename[sizeof "./images/file00000.png"];

    image_size_t sz; 
    sz.width=nx;
    sz.height=ny;

    //make mesh
    double z[W][H];

    //Velocity
    double v[W][H];

    //Accelleration
    double a[W][H];

    //output image
    char * o_img = (char *) malloc(sz.width*sz.height*sizeof(char));
    char ** output = (char **) malloc(sz.height * sizeof(char*));
    for (int r=0; r<sz.height; r++)
        output[r] = &o_img[r*sz.width];

    max=10.0;
    min=0.0;
    dx = (max-min)/(double)(nx-1);
    dy = (max-min)/(double)(ny-1);
    
    tmax=20.0;
    dt= (tmax-0.0)/(double)(nt-1);

    double x,y; 
    for (r=0;r<ny;r++)  {
        for (c=0;c<nx;c++)  {
        x = min+(double)c*dx;
        y = min+(double)r*dy;
            z[r][c] = exp(-(sqrt((x-5.0)*(x-5.0)+(y-5.0)*(y-5.0))));
            a[r][c] = 0.0;
            v[r][c] = 0.0;
}
    }
    
    dx2inv=1.0/(dx*dx);
    dy2inv=1.0/(dy*dy);


    //We have initialized everthing, so now we want to do some memalloc and memcopying
    CUDA_CALL(hipMalloc((void **)&d_z, nx*ny*sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_v, nx*ny*sizeof(double)));
    CUDA_CALL(hipMalloc((void **)&d_a, nx*ny*sizeof(double)));

    CUDA_CALL(hipMemcpy(d_z,z,nx*sizeof(double)*ny,hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_v,v,nx*sizeof(double)*ny,hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_a,a,nx*sizeof(double)*ny,hipMemcpyHostToDevice));

    //make dim3's
    int block_size = 32;
    int nblocks_x = nx/block_size;
    int nblocks_y = ny/block_size;
    dim3 dimGrid(nblocks_x, nblocks_y, 1);
    dim3 dimBlock(block_size, block_size, 1);



    for(it=0;it<nt-1;it++) {
    //printf("%d\n",it);
        
            calcAccel<<<dimGrid,dimBlock>>>(dx2inv, dy2inv, nx, ny, d_z, d_a);
            calcVelAndPos<<<dimGrid, dimBlock>>>(dt, nx, ny, d_z, d_v, d_a);

    if (it % 100 ==0)
    {
            CUDA_CALL(hipMemcpy(z, d_z, nx*ny*sizeof(double), hipMemcpyDeviceToHost));
            double mx,mn;
            mx = -999999;
            mn = 999999;
            for(r=0;r<ny;r++)
                for(c=0;c<nx;c++){
                   mx = max(mx, z[r][c]);
                   mn = min(mn, z[r][c]);
            }
            for(r=0;r<ny;r++)
                for(c=0;c<nx;c++)
                    output[r][c] = (char) round((z[r][c]-mn)/(mx-mn)*255);

            sprintf(filename, "./images/file%05d.png", frame);
            printf("Writing %s\n",filename);    
            write_png_file(filename,(unsigned char *) o_img,sz);
        frame+=1;
        }

    }
    
    double mx,mn;
    mx = -999999;
    mn = 999999;
    for(r=0;r<ny;r++)
        for(c=0;c<nx;c++){
       mx = max(mx, z[r][c]);
       mn = min(mn, z[r][c]);
        }

    printf("%f, %f\n", mn,mx);

    for(r=0;r<ny;r++)
        for(c=0;c<nx;c++){  
       output[r][c] = (char) round((z[r][c]-mn)/(mx-mn)*255);  
    }

    sprintf(filename, "./images/file%05d.png", it);
    printf("Writing %s\n",filename);    
    //Write out output image using 1D serial pointer
    write_png_file(filename,(unsigned char *) o_img,sz);
    return 0;
}
